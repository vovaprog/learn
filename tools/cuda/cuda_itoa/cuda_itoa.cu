
#include <hip/hip_runtime.h>
#include <stdio.h>

struct StringData {
	char str[11];
};

unsigned int *devDataInput;
StringData *devStringDataOutput;
unsigned int dataCount;

bool cdInit(unsigned int dataCountArg)
{
	dataCount = dataCountArg;
	
	hipError_t cudaStatus;
    
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice error: %d\r\n", (int)cudaStatus);
        return false;
    }

    cudaStatus = hipMalloc((void**)&devDataInput, dataCount * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMalloc error: %d\r\n", (int)cudaStatus);
		return false;
	}

    cudaStatus = hipMalloc((void**)&devStringDataOutput, dataCount * sizeof(StringData));
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMalloc error: %d\r\n", (int)cudaStatus);
		return false;
	}
	
	return true;
}


__device__ void uintToStringDevice(unsigned int x, char *output)
{
	const int bufSize = 11;
	char buf[bufSize];
	buf[bufSize - 1] = 0;
	int ind = bufSize - 2;
	
	do {
		buf[ind] = '0' + x % 10;
		x /= 10;
		--ind;
	} while (x != 0);

	++ind;

	int i;
	for(i = 0; buf[ind] != 0; ++i, ++ind)
	{
		output[i] = buf[ind];
	}
	output[i] = 0;
}


__global__ void cdItoaDevice(unsigned int *dataInput, StringData *stringDataOutput)
{
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	
	uintToStringDevice(dataInput[threadIndex], stringDataOutput[threadIndex].str);	
}


bool cdItoa(unsigned int *dataInput, StringData *stringDataOutput)
{
	hipError_t cudaStatus=hipSuccess;
	
    cudaStatus = hipMemcpy(devDataInput, dataInput, dataCount * sizeof(unsigned int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMemcpy error: %d\r\n", (int)cudaStatus);
		return false;
	}
	
	int numberOfBlocks = 1024;
	int threadsPerBlock = dataCount / numberOfBlocks;

	cdItoaDevice<<<numberOfBlocks, threadsPerBlock>>>(devDataInput, devStringDataOutput);
	
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "cuda error: %s\r\n", hipGetErrorString(cudaStatus));
        return false;
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize error: %d %s\r\n", (int)cudaStatus, hipGetErrorString(cudaStatus));
        return false;
    }	

    cudaStatus = hipMemcpy(stringDataOutput, devStringDataOutput, dataCount * sizeof(StringData), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMemcpy error: %d\r\n", (int)cudaStatus);
		return false;
	}
	
    return true;
}

