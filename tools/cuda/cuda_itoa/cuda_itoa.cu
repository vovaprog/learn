
#include <hip/hip_runtime.h>
#include <stdio.h>

struct StringData {
	char str[11];
};

unsigned int *devDataInput;
StringData *devStringDataOutput;
unsigned int dataCount;

template< typename T >
void check(T result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n",
                file, line, static_cast<unsigned int>(result), hipGetErrorString(result), func);        
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}

#define checkCudaErrors(val)           check ( (val), #val, __FILE__, __LINE__ )

bool cdInit(unsigned int dataCountArg, void **hostInputMemory, void **hostOutputMemory, bool allocPinnedMemory)
{
	dataCount = dataCountArg;
	
	hipError_t cudaStatus;
    
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice error: %d\r\n", (int)cudaStatus);
        return false;
    }

    if(allocPinnedMemory)
    {
    	checkCudaErrors(hipHostAlloc((void **)hostInputMemory, dataCount * sizeof(unsigned int), hipHostMallocWriteCombined));
    	checkCudaErrors(hipHostAlloc((void **)hostOutputMemory, dataCount * sizeof(StringData), 0));
    }
    else
    {
    	*hostInputMemory = (void *)malloc(dataCount * sizeof(unsigned int));
    	*hostOutputMemory = (void *)malloc(dataCount * sizeof(StringData));
    }
	    
	
    cudaStatus = hipMalloc((void**)&devDataInput, dataCount * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMalloc error: %d\r\n", (int)cudaStatus);
		return false;
	}

    cudaStatus = hipMalloc((void**)&devStringDataOutput, dataCount * sizeof(StringData));
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMalloc error: %d\r\n", (int)cudaStatus);
		return false;
	}
	
	return true;
}


__device__ void uintToStringDevice(unsigned int x, char *output)
{
	const int bufSize = 11;
	char buf[bufSize];
	buf[bufSize - 1] = 0;
	int ind = bufSize - 2;
	
	do {
		buf[ind] = '0' + x % 10;
		x /= 10;
		--ind;
	} while (x != 0);

	++ind;
	
	int i;
	for(i = 0; buf[ind] != 0; ++i, ++ind)
	{
		output[i] = buf[ind];
	}
	output[i] = 0;
}


__global__ void cdItoaDevice(unsigned int *dataInput, StringData *stringDataOutput)
{
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	
	uintToStringDevice(dataInput[threadIndex], stringDataOutput[threadIndex].str);	
}


bool cdItoa(unsigned int *dataInput, StringData *stringDataOutput)
{
	hipError_t cudaStatus=hipSuccess;
	
    cudaStatus = hipMemcpy(devDataInput, dataInput, dataCount * sizeof(unsigned int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMemcpy error: %d\r\n", (int)cudaStatus);
		return false;
	}
	
	int numberOfBlocks = 1024;
	int threadsPerBlock = dataCount / numberOfBlocks;

	cdItoaDevice<<<numberOfBlocks, threadsPerBlock>>>(devDataInput, devStringDataOutput);
	
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "cuda error: %s\r\n", hipGetErrorString(cudaStatus));
        return false;
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize error: %d %s\r\n", (int)cudaStatus, hipGetErrorString(cudaStatus));
        return false;
    }	

    cudaStatus = hipMemcpy(stringDataOutput, devStringDataOutput, dataCount * sizeof(StringData), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMemcpy error: %d\r\n", (int)cudaStatus);
		return false;
	}
	
    return true;
}

