#include "hip/hip_runtime.h"
#include <stdio.h>

#include "data.h"

VenueData *devVenueInput;
ClientData *devClientOutput;
unsigned int dataCount;

bool cdInit(unsigned int dataCountArg)
{
	dataCount = dataCountArg;
	
	hipError_t cudaStatus;
    
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice error: %d\r\n", (int)cudaStatus);
        return false;
    }

    cudaStatus = hipMalloc((void**)&devVenueInput, dataCount * sizeof(VenueData));
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMalloc error: %d\r\n", (int)cudaStatus);
		return false;
	}

    cudaStatus = hipMalloc((void**)&devClientOutput, dataCount * sizeof(ClientData));
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMalloc error: %d\r\n", (int)cudaStatus);
		return false;
	}
	
	return true;
}


__device__ void llToStringDevice(unsigned long long int x, char *output)
{
	const int bufSize = 30;
	char buf[bufSize];
	buf[bufSize - 1] = 0;
	int ind = bufSize - 2;
	
	do {
		buf[ind] = '0' + x % 10;
		x /= 10;
		--ind;
		
	} while (x != 0);
	
	for(int i=0;buf[ind]!=0;++i, ++ind)
	{
		output[i] = buf[ind];
	}
}


__device__ void translateVenueToClientDevice(const VenueData &v, ClientData &c)
{
	//sprintf(c.id, "%llu", (unsigned long long int)v.id);
	llToStringDevice(v.id, c.id);
	c.price = v.price * 100;
	c.quantity = v.quantity;
	memcpy(c.userName, v.userName, sizeof(v.userName));
		
	switch(v.orderType) {
	case VenueOrderType::Market: c.orderType = ClientOrderType::Market; break;
	case VenueOrderType::Limit: c.orderType = ClientOrderType::Limit; break;
	case VenueOrderType::Pegged: c.orderType = ClientOrderType::Pegged; break;
	}
}


__global__ void cdTranslateDevice(VenueData *devVenueInput, ClientData *devClientOutput)
{
	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	
	translateVenueToClientDevice(devVenueInput[threadIndex], devClientOutput[threadIndex]);	
}


bool cdTranslateVenueToClient(VenueData *venueInput, ClientData *clientOutput)
{
	hipError_t cudaStatus=hipSuccess;
	
    cudaStatus = hipMemcpy(devVenueInput, venueInput, dataCount * sizeof(VenueData), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMemcpy error: %d\r\n", (int)cudaStatus);
		return false;
	}
	
	int numberOfBlocks = 512;
	int threadsPerBlock = dataCount / numberOfBlocks;

	cdTranslateDevice<<<numberOfBlocks, threadsPerBlock>>>(devVenueInput, devClientOutput);
	
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "cuda error: %s\r\n", hipGetErrorString(cudaStatus));
        return false;
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize error: %d %s\r\n", (int)cudaStatus, hipGetErrorString(cudaStatus));
        return false;
    }	

    cudaStatus = hipMemcpy(clientOutput, devClientOutput, dataCount * sizeof(ClientData), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMemcpy error: %d\r\n", (int)cudaStatus);
		return false;
	}
	
    return true;
}

bool cdTranslateVenueToClientStart(VenueData *venueInput)
{
	hipError_t cudaStatus=hipSuccess;
	
    cudaStatus = hipMemcpy(devVenueInput, venueInput, dataCount * sizeof(VenueData), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMemcpy error: %d\r\n", (int)cudaStatus);
		return false;
	}
	
	int numberOfBlocks = 512;
	int threadsPerBlock = dataCount / numberOfBlocks;

	cdTranslateDevice<<<numberOfBlocks, threadsPerBlock>>>(devVenueInput, devClientOutput);
	
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "cuda error: %s\r\n", hipGetErrorString(cudaStatus));
        return false;
    }
    
    return true;
}

bool cdTranslateVenueToClientEnd(ClientData *clientOutput)
{    
	hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize error: %d %s\r\n", (int)cudaStatus, hipGetErrorString(cudaStatus));
        return false;
    }	

    cudaStatus = hipMemcpy(clientOutput, devClientOutput, dataCount * sizeof(ClientData), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMemcpy error: %d\r\n", (int)cudaStatus);
		return false;
	}
	
    return true;
}


